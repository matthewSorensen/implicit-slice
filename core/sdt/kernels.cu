
#include <hip/hip_runtime.h>
__device__ int sgn(float x){
  if(x == 0.0){
    return 0;
  }else if(x < 0.0){
    return -1;
  }
  return 1;
}

__device__ int square(int x){
  return x*x;
}

__global__ void extract_zeros(float* implicit,int* output, int width,int height){
  // We are going with 16*16 blocks, so we need (16 + 2) * 16 bytes per thread
  __shared__ char allsigns[288];
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int i = x + y * width;
  int replace = max(width,height);
  replace *= replace;

  if(width <= x || height <= y){ 
    __syncthreads();
    return;
  }

  int here = sgn(implicit[i]);

  char* signs = &(allsigns[18 * threadIdx.y]);
  signs[threadIdx.x + 1] = here;
  
  if(x == 0){
    signs[0] = here;
  }else if(x == (width -1)){
    signs[threadIdx.x + 2] = here;
  }else if(threadIdx.x == 0){
    signs[0] = sgn(implicit[i-1]);
  } else if(threadIdx.x == (blockDim.x -1)){
    signs[17] = sgn(implicit[i+1]);
  }  
  __syncthreads();
  if((here == 1) && ((signs[threadIdx.x] == -1) || (signs[threadIdx.x + 2] == -1))){
    here = 0;
  }
  output[i] = here * replace;
}

#define MAXBLOCK 512

__global__ void edt_pass(int* samples, const int width, const int height, const int dim){

  __shared__ int coeffs[MAXBLOCK];
  __shared__ int verts[MAXBLOCK];

  // this requires that the thread size is (length of data,1)
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  int* sample = &(samples[x + y * width]);
  int original = *sample;
  int out = abs(original);

  int frame;
  int pos;
  int size;
  if(dim){
    frame = y & ~1;
    pos = y;
    size = height;
  }else{
    frame = x & ~1;
    pos = x;
    size = width;
  }
 
  // Perform the first set of reductions
  coeffs[pos] = out;
  __syncthreads();
  int otherindex = pos ^ 1;
  int otherdata = coeffs[otherindex];

  if(out > otherdata){
    coeffs[pos] = otherdata + 1;
    verts[pos] = otherindex;
  }else{
    verts[pos] = pos;
  }
  
  __syncthreads();  


  int mask = 3; 

  while(size > 0){
    size >>= 1;

    int base = frame & ~3;
    int dest = base >> 1;

    int offset = base ^ frame;
    int half = offset >> 1;
    offset = offset | half;

    int par = pos & mask;

    int low = square(pos - verts[base + 1]) + coeffs[base + 1];
    int high = square(pos - verts[base + 2]) + coeffs[base + 2];
    int extreme = square(pos - verts[base + offset]) + coeffs[base + offset];

    out = min(out,min(high,min(low,extreme)));
    
    if(par == 0 || par == mask){
      if(high < extreme || low < extreme){
	offset = (offset + 2) & 3;
      }
      
      int vertex = verts[base + offset];
      int coefficient = coeffs[base + offset];
      
      __syncthreads();
      coeffs[dest + half] = coefficient;
      verts[dest + half] = vertex;
      __syncthreads();
    }else{
      __syncthreads();
      __syncthreads();
    }
    
    frame = dest;
    mask = (mask << 1) + 1;    
  }
  
  if(original < 0) out = -1 * out;
  
  *sample = out;  
}
