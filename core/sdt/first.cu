
#include <hip/hip_runtime.h>

__device__ int sgn(const float f){
  if(f == 0.0)
    return 0;
  if(signbit(f))
    return -1;
  return 1;
}

__device__ float safe_copysign(const float x, const float y){
  // returns x with the sign of y
  if(x == 0.0 || y == 0.0){
    return 0.0;
  }
  return copysign(x,y);
}

__device__ float square(const float x){
  return x * x;
}

__global__ void horizontal(float* sample, float * dest, const int width, const int pitch, const int height){
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  if(height <= x) return;
  sample = sample + x * pitch;
  dest = dest + x * pitch;

  int psgn = sgn(sample[0]);
  int last = -1;

  for(int i = 0; i < width; i++){
    int ssgn = sgn(sample[i]);

    if(ssgn != psgn){
      int upper = i;
      if(psgn == 0)
	upper--;
   
      if(last == -1){
	for(int x = 0; x < upper; x++){
	  dest[x] = safe_copysign(square(x - upper), sample[x]);
	}
      } else {
	for(int x = last; x <= upper; x++){
	  dest[x] = safe_copysign(square(fminf(x - last,upper - x)), sample[x]);
	}
      }
      last = upper;
    } else if (ssgn == 0){
      last = i + 1;
    }
    psgn = ssgn;
  }
  if(last == -1){
    float val = safe_copysign(width * width, sample[0]);
    for(int i = 0; i < width; i++){
      dest[i] = val;
    }
  } else {
    for(int i = last; i < width; i++){
      dest[i] = safe_copysign(square(i - last), sample[i]);
    }
  }
}
