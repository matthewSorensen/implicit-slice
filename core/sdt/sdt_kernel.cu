
#include <hip/hip_runtime.h>

__device__ int sgn(const float f){
  if(f == 0.0)
    return 0;
  if(f < 0.0){
    return -1;
  }
  return 1;
  //  if(signbit(f))
  //   return -1;
  //return 1;
}

__device__ float safe_copysign(const float x, const float y){
  // returns x with the sign of y
  if(x == 0.0 || y == 0.0){
    return 0.0;
  }
  return copysignf(x,y);
}

__device__ float square(const float x){
  return x * x;
}

__global__ void implicit_first_pass(float* sample, const int width, const int height){
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  if(height <= x) return;
  sample = &(sample[x]);
  
  int psgn = sgn(sample[0]);
  int last = -1;

  for(int i = 0; i < height; i++){
    int ssgn = sgn(sample[width * i]);

    if(ssgn != psgn){
      int upper = i;
      if(psgn == 0)
	upper--;   
      if(last == -1){
	for(int x = 0; x < height; x++){
	  sample[width * x] = safe_copysign(square(x - upper), sample[width * x]);
	}
      } else {
	for(int x = last; x <= height; x++){
	  sample[width * x] = safe_copysign(square(fminf(x - last,upper - x)), sample[width * x]);
	}
      }
      last = upper;
    } else if (ssgn == 0){
      last = i + 1;
    }
    psgn = ssgn;
  }
  if(last == -1){
    float val = safe_copysign(square(fmaxf(width, height) * 2), sample[0]);
    for(int i = 0; i < height; i++){
      sample[width * i] = val;
    }
  } else {
    for(int i = last; i < height; i++){
      sample[width * i] = safe_copysign(square(i - last), sample[width * i]);
    }
  }
  
}

__global__ void voxel_first_pass(float* sample, const int width, const int height){
 const int x = blockIdx.x * blockDim.x + threadIdx.x;
  if(height <= x) return;
  sample = &(sample[x]);
  
  int psgn = sgn(sample[0]);
  int last = -1;

  for(int i = 0; i < height; i++){
    int ssgn = sgn(sample[width * i]);

    if(ssgn != psgn){
      int upper = i;
      if(last == -1){
	for(int x = 0; x < height; x++){
	  sample[width * x] = safe_copysign(square(x - upper), sample[width * x]);
	}
      } else {
	for(int x = last; x <= height; x++){
	  sample[width * x] = safe_copysign(square(fminf(x - last,upper - x)), sample[width * x]);
	}
      }
      last = upper;
    }
    psgn = ssgn;
  }
  if(last == -1){
    float val = safe_copysign(square(fmaxf(width, height) * 2), sample[0]);
    for(int i = 0; i < height; i++){
      sample[width * i] = val;
    }
  } else {
    for(int i = last; i < height; i++){
      sample[width * i] = safe_copysign(square(i - last), sample[width * i]);
    }
  }
}

#define INF 0x7f800000
#define NINF 0xff800000 


__global__ void second_pass(float* twod, float* bounds,int* verts, float* out,int width, int height){
  const int y = blockIdx.x * blockDim.x + threadIdx.x;
  if(height <= y) return;
  twod = &(twod[y * width]);
  out = &(out[y * width]);
  bounds = &(bounds[y * (width + 1)]);
  verts = &(verts[y * width]);
  
  int k = 0;
  bounds[0] = NINF;
  bounds[1] = INF;
  verts[0] = 0;
  
  for(int q = 1; q < width; q++){
    float sample = fabsf(twod[q]);
    float ss = sample + square(q);
    float inter = ss - fabsf(twod[verts[k]]) - square(verts[k]);
    
    inter *= 0.5 / (q - verts[k]);
    
    while(inter <= bounds[k]){
      verts[k] = 0;
      k--;
      inter = ss - fabsf(twod[verts[k]]) - square(verts[k]);
      inter *= 0.5 / (q - verts[k]);
    }
    
    k++;
    verts[k]  = q;
    bounds[k] = inter;
    bounds[k+1] = INF;
  }
  
  k = 0;
  for(int q = 0; q < width; q++){
    while(bounds[k+1] < q)
      k++;
    out[q] = square(q - verts[k]) + fabsf(twod[verts[k]]);
    }


}

__global__ void sign_and_sqrt(float* signs, float* values, float* output, int width, int height){
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(width <= x || height <= y) return;
  int i = x + width * y;

  output[i] = safe_copysign(signs[i],sqrtf(values[i]));
}

